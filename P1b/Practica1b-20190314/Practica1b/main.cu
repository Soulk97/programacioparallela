#include <iostream>
#include <hip/hip_runtime.h>

#define WIDTH 3833
#define HEIGHT 2160

bool checkResults(uchar4* rgba, uchar3* bgr, int size) {

    bool correct = true;

    for (int i=0; i < size; ++i) {
        correct &= rgba[i].x == bgr[i].z;
        correct &= rgba[i].y == bgr[i].y;
        correct &= rgba[i].z == bgr[i].x;
        correct &= rgba[i].w == 255;
    }

    return correct;
}

__global__ void convertBGR2RGBA(uchar3 *bgr, uchar4* rgba, int width, int height) {

	int position = 0;// 0 is not correct. Compute each thread position;

	// Protection to avoid segmentation fault
	if (position < width * height) {	
		rgba[position].x == bgr[position].z;
		rgba[position].y == bgr[position].y;
		rgba[position].z == bgr[position].x;
		rgba[position].w == 255;
	}
}

int main() {

    uchar3 *h_bgr, *d_bgr;
    uchar4 *h_rgba, *d_rgba;

    int bar_widht = HEIGHT/3;

    // Alloc and generate BGR bars.
    h_bgr = (uchar3*)malloc(sizeof(uchar3)*WIDTH*HEIGHT);
    for (int i=0; i < WIDTH * HEIGHT; ++i) {
        if (i < bar_widht) { h_bgr[i] = { 255, 0, 0 }; }
        else if (i < bar_widht*2) { h_bgr[i] = { 0, 255, 0 }; }
        else { h_bgr[i] = { 0, 0, 255 }; }
    }

    // Alloc RGBA pointers
    h_rgba = (uchar4*)malloc(sizeof(uchar4)*WIDTH*HEIGHT);

    // Alloc gpu pointers
    hipError_t error = hipMalloc(&d_bgr, sizeof(uchar3) * WIDTH * HEIGHT);
    if (error != hipSuccess) {
	std::cout << "Error in hipMalloc" << std::endl;
    }

    error = hipMalloc(&d_rgba, sizeof(uchar4) * WIDTH * HEIGHT);
    if (error != hipSuccess) {
	std::cout << "Error in hipMalloc" << std::endl;
    }
    
    // Copy data to GPU
    error = hipMemcpy(d_bgr, h_bgr, sizeof(uchar3) * WIDTH * HEIGHT, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
	std::cout << "Error in hipMemcpy" << std::endl;
    }

    dim3 block(512, 1, 1);
    dim3 grid(ceil(WIDTH*HEIGHT/(float)block.x), 1, 1);
    convertBGR2RGBA<<<grid, block, 0, 0>>>(d_bgr, d_rgba, WIDTH, HEIGHT);

    error = hipMemcpy(h_rgba, d_rgba, sizeof(uchar4) * WIDTH * HEIGHT, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
	std::cout << "Error in hipMemcpy" << std::endl;
    }

    bool ok = checkResults(h_rgba, h_bgr, WIDTH*HEIGHT);

    if (ok) {
        std::cout << "Executed!! Results OK." << std::endl;
    } else {
        std::cout << "Executed!! Results NOT OK." << std::endl;
    }

    return 0;

}
